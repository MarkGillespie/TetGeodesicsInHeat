#include "hip/hip_runtime.h"
#include "cg.cuh"

#define NTHREAD 128 
#define NBLOCK 1024

// Computes out = (M + tL)p
__global__ void computeAp(double *out, double *p, double *cotans, int* neighbors, double* m, double t, int meshStride, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(int i = index; i < n; i += stride){
        out[i] = m[i] * p[i];
        for (int iN = 0; iN < meshStride; ++iN) {
            int neighbor = neighbors[i * meshStride + iN];
            double weight = cotans[i * meshStride + iN];
            out[i] += t * weight * (p[i] - p[neighbor]);
        }
    }
}

// Computes out = (M + tL)p
__global__ void computeApCSR(double *out, double *p, double *cotans, int* neighbors, double* m, int* end, double t, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = m[i] * p[i];
        for (int iN = end[i-1]; iN < end[i]; ++iN) {
            int neighbor = neighbors[iN];
            double weight = cotans[iN];
            out[i] += t * weight * (p[i] - p[neighbor]);
        }
    }
}

// Computes out = (M + tL)p
// TODO: make sure there are enough blocks
__global__ void computeApClusteredCSR(double *out,
                                      double *global_p,
                                      double *cotans,
                                      int* neighbors,
                                      int* vertex_end,
                                      int* local_indices,
                                      int* local_ind_end, 
                                      int* block_size,
                                      double* m,
                                      double t,
                                      int n) {

    extern __shared__ double p[];

    int nEntries = local_ind_end[blockIdx.x + 1] - local_ind_end[blockIdx.x];
    for (int i = threadIdx.x; i < nEntries; i += blockDim.x) {
        p[i] = global_p[local_indices[local_ind_end[blockIdx.x] + i]];
    }

    /*
    __syncthreads();

    if (threadIdx.x < block_size[blockIdx.x]) {
        int i = local_indices[local_ind_end[blockIdx.x] + threadIdx.x];
        //out[i] = m[i] * p[threadIdx.x];
        //for (int iN = vertex_end[i-1]; iN < vertex_end[i]; ++iN) {
            //int neighbor = neighbors[iN];
            //double weight = cotans[iN];
            //out[i] += t * weight * (p[threadIdx.x] - p[neighbor]);
        //}
        //out[i] = p[threadIdx.x];
        out[i] = 7;
    }
*/
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        out[0] = 7;
    }
}

// Computes out = a-b
__global__ void vector_sub(double *out, double *a, double *b, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i] - b[i];
  }
}

// Copies a into out
__global__ void vector_cpy(double *out, double *a, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i];
  }
}

// Computes out = num / denom
__global__ void div(double *out, double *num, double *denom) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0) {
      *out = *num / *denom;
  }
}

// x += alpha p
// r -= alpha Ap
__global__ void update_x_r(double* x, double* r, double* alpha, double* p, double* Ap, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    x[i] += alpha[0] * p[i];
    r[i] -= alpha[0] * Ap[i];
  }
}

// Computes p = beta * p + r
__global__ void update_p(double* p, double *beta, double *r, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    p[i] = beta[0] * p[i] + r[i];
  }
}

// For each vertex, returns a map which maps the indices of the vertex's neighbors to the
// entries of the laplacian corresponding to those edges
std::vector<std::unordered_map<size_t, double>> edgeWeights(const TetMesh& mesh) {
    std::vector<std::unordered_map<size_t, double>> weights;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        std::unordered_map<size_t, double> vWeights;
        weights.push_back(vWeights);
    }

    for (size_t iPE = 0; iPE < mesh.edges.size(); ++iPE) {
        PartialEdge pe = mesh.edges[iPE];
        size_t vSrc    = pe.src;
        size_t vDst    = pe.dst;

        double weight = mesh.partialEdgeCotanWeights[iPE];
        weights[vSrc][vDst] += weight;
        weights[vDst][vSrc] += weight;
    }

    return weights;
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
int  cgSolve(Eigen::VectorXd& xOut, const Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose) {
    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors;
    int N = bVec.size();

    int maxDegree = 0;
    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        maxDegree = std::max(maxDegree, (int) weights[iV].size());
    }

    // Allocate host memory
    x         = (double*) malloc(sizeof(double) * N);
    b         = (double*) malloc(sizeof(double) * N);
    r         = (double*) malloc(sizeof(double) * N);
    m         = (double*) malloc(sizeof(double) * N);
    cotans    = (double*) malloc(sizeof(double) * N * maxDegree);
    neighbors = (int*  ) malloc(sizeof(int  ) * N * maxDegree);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[i];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[i];
    }
    if (t < 0) t = 1;

    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        size_t neighborCount = 0;
        for (std::pair<size_t, double> elem : weights[iV]) {
            neighbors[iV * maxDegree + neighborCount] = elem.first;
            cotans[iV * maxDegree + neighborCount] = elem.second;
            ++neighborCount;
        }

        // Fill in the remaining slots with zeros
        for (size_t iN = neighborCount; iN < maxDegree; ++iN) {
            neighbors[iV * maxDegree + iN] = iV;
            cotans[iV * maxDegree + iN] = 0;
        }
    }

    //printf("max degree: %d\n", maxDegree);
    Eigen::SparseMatrix<double> L    = mesh.weakLaplacian();
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        for (size_t iN = 0; iN < maxDegree; ++iN) {
            size_t jV = neighbors[iV * maxDegree + iN];
            if (jV == iV) continue;

            double mat = L.coeffRef(iV, jV);
            double arr = cotans[iV * maxDegree + iN];
            if (abs(mat + arr) >= 1e-7) {
                printf("ERROR: matrix is %f\tarray is %f\terror is %.10e\n",
                        mat, arr, abs(mat + arr));
            }
        }
    }

    // Allocate device memory
    hipMalloc((void**)&d_x,         sizeof(double) * N);
    hipMalloc((void**)&d_b,         sizeof(double) * N);
    hipMalloc((void**)&d_Ap,        sizeof(double) * N);
    hipMalloc((void**)&d_p,         sizeof(double) * N);
    hipMalloc((void**)&d_r,         sizeof(double) * N);
    hipMalloc((void**)&d_m,         sizeof(double) * N);
    hipMalloc((void**)&d_neighbors, sizeof(int   ) * N * maxDegree);
    hipMalloc((void**)&d_cotans,    sizeof(double) * N * maxDegree);
    hipMalloc((void**)&d_alpha,     sizeof(double) * 1);
    hipMalloc((void**)&d_beta,      sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,       sizeof(double) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_x,         x,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_r,         r,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_b,         b,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_m,         m,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors, sizeof(int   ) * N * maxDegree, hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,    cotans,    sizeof(double) * N * maxDegree, hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE); 

    computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_x, d_cotans, d_neighbors, d_m, t, maxDegree, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
         computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_p, d_cotans, d_neighbors, d_m, t, maxDegree, N);
         hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
         div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
         update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
         hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
         div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
         update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
         vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 300);
    }
    hipblasDestroy(handle);

    if (norm >= tol)
        printf("timed out semidense :'(");

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[i] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_m);
    hipFree(d_neighbors);
    hipFree(d_cotans);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_pAp);

    // Deallocate host memory
    free(x);
    free(b);
    free(r);
    free(m);
    free(cotans);
    free(neighbors);

    return iter * substeps;
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
// Stores matrix in CSR format
int cgSolveCSR(Eigen::VectorXd& xOut, const Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose,
               std::vector<size_t> vertexPermutation) {
    std::vector<size_t> invPerm(vertexPermutation.size(), 0);
    for (size_t i = 0; i < vertexPermutation.size(); ++i) {
        invPerm[vertexPermutation[i]] = i;
    }

    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors, *end, *d_end;
    int N = bVec.size();

    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    int nEdges = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        nEdges += weights[iV].size();
    }

    // Allocate host memory
    x         = (double*) malloc(sizeof(double) * N);
    b         = (double*) malloc(sizeof(double) * N);
    r         = (double*) malloc(sizeof(double) * N);
    m         = (double*) malloc(sizeof(double) * N);
    cotans    = (double*) malloc(sizeof(double) * nEdges);
    neighbors = (int*   ) malloc(sizeof(int   ) * nEdges);
    end = (int*   ) malloc(sizeof(int   ) * (N+1));

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[vertexPermutation[i]];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[vertexPermutation[i]];
    }
    if (t < 0) t = 1;

    int pos = 0;
    end[0] = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        for (std::pair<size_t, double> elem : weights[vertexPermutation[iV]]) {
            neighbors[pos] = invPerm[elem.first];
            cotans[pos] = elem.second;
            pos += 1;
        }
        end[iV] = pos;
    }

    // Allocate device memory
    hipMalloc((void**)&d_x,         sizeof(double) * N);
    hipMalloc((void**)&d_b,         sizeof(double) * N);
    hipMalloc((void**)&d_Ap,        sizeof(double) * N);
    hipMalloc((void**)&d_p,         sizeof(double) * N);
    hipMalloc((void**)&d_r,         sizeof(double) * N);
    hipMalloc((void**)&d_m,         sizeof(double) * N);
    hipMalloc((void**)&d_neighbors, sizeof(int   ) * nEdges);
    hipMalloc((void**)&d_cotans,    sizeof(double) * nEdges);
    hipMalloc((void**)&d_end,       sizeof(int   ) * (N+1));
    hipMalloc((void**)&d_alpha,     sizeof(double) * 1);
    hipMalloc((void**)&d_beta,      sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,       sizeof(double) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_x,         x,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_r,         r,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_b,         b,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_m,         m,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors, sizeof(int   ) * nEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_end,       end,       sizeof(int   ) * (N+1),  hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,    cotans,    sizeof(double) * nEdges, hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    computeApCSR<<<NBLOCK,NTHREAD>>>(d_Ap, d_x, d_cotans, d_neighbors, d_m, d_end, t, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
        computeApCSR<<<NBLOCK,NTHREAD>>>(d_Ap, d_p, d_cotans, d_neighbors, d_m, d_end, t, N);
        hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
        div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
        update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
        hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
        div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
        update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
        vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 1000);
    }
    if (norm >= tol)
        printf("timed out csr :'(");
    hipblasDestroy(handle);

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[vertexPermutation[i]] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_m);
    hipFree(d_neighbors);
    hipFree(d_cotans);
    hipFree(d_end);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_pAp);


    // Deallocate host memory
    free(x);
    free(b);
    free(r);
    free(m);
    free(cotans);
    free(neighbors);

    return iter * substeps;
}

int cgSolveCSR(Eigen::VectorXd& xOut, const Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose, bool degreeSort) {
    std::vector<size_t> perm;
    perm.reserve(mesh.vertices.size());
    if (degreeSort) {
        std::vector<std::pair<size_t, size_t>> degreeIndexPairs;
        degreeIndexPairs.reserve(mesh.vertices.size());
        std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
        for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
            degreeIndexPairs.push_back(std::make_pair(weights[iV].size(), iV));
        }
        auto cmp = [](std::pair<size_t, size_t>a, std::pair<size_t, size_t> b) {return a.first < b.first;};
        std::sort(degreeIndexPairs.begin(), degreeIndexPairs.end(), cmp);
        for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
            perm.push_back(degreeIndexPairs[iV].second);
        }
    } else {
        for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) perm.push_back(iV);
    }
    return cgSolveCSR(xOut, bVec, mesh, tol, t, verbose, perm);
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
// Stores matrix in CSR format
int cgSolveClusteredCSR(Eigen::VectorXd& xOut, const Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose) {
    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors, *end, *d_end, *local_indices, *d_local_indices, *local_ind_end, *d_local_ind_end, *block_size, *d_block_size;
    int N = bVec.size();

    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    int nEdges = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        nEdges += weights[iV].size();
    }

    std::vector<std::vector<size_t>> clusters, clusterNeighbors;
    std::tie(clusters, clusterNeighbors) = clusterAndNeighbors(mesh, NTHREAD);
    size_t nClusters = clusters.size();
    size_t nVerticesWithDupes = 0;
    for (size_t iC = 0; iC < nClusters; ++iC) {
        nVerticesWithDupes += clusters[iC].size();
        nVerticesWithDupes += clusterNeighbors[iC].size();
    }

    // Allocate host memory
    x         = (double*) malloc(sizeof(double) * N);
    b         = (double*) malloc(sizeof(double) * N);
    r         = (double*) malloc(sizeof(double) * N);
    m         = (double*) malloc(sizeof(double) * N);
    cotans    = (double*) malloc(sizeof(double) * nEdges);
    neighbors = (int*   ) malloc(sizeof(int   ) * nEdges);
    end       = (int*   ) malloc(sizeof(int   ) * (N+1));
    local_indices  = (int*) malloc(sizeof(int) * nVerticesWithDupes);
    local_ind_end  = (int*) malloc(sizeof(int) * (NBLOCK+1));
    block_size     = (int*) malloc(sizeof(int) * NBLOCK);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[i];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[i];
    }
    if (t < 0) t = 1;

    std::vector<size_t> vertexCluster(N, 0);
    std::vector<std::vector<int>> clusterLocalIndices(N, std::vector<int>(N, -1));

    int clusterPos = 0;
    local_ind_end[0] = 0;
    size_t MAX_CLUSTER_SIZE = 0;
    for (size_t iC = 0; iC < nClusters; ++iC) {
        for (size_t iV = 0; iV < clusters[iC].size(); ++iV) {
            vertexCluster[clusters[iC][iV]] = iC;
            local_indices[clusterPos] = clusters[iC][iV];
            clusterLocalIndices[iC][clusters[iC][iV]] = iV;
            clusterPos += 1;
        }
        for (size_t iV = 0; iV < clusterNeighbors[iC].size(); ++iV) {
            local_indices[clusterPos] = clusterNeighbors[iC][iV];
            clusterLocalIndices[iC][clusterNeighbors[iC][iV]] = clusters[iC].size() + iV;
            clusterPos += 1;
        }
        local_ind_end[iC+1] = clusterPos;
        block_size[iC] = clusters[iC].size();
        MAX_CLUSTER_SIZE = std::max(MAX_CLUSTER_SIZE, clusters[iC].size() + clusterNeighbors[iC].size());
    }
    for (size_t extra = nClusters; extra < NBLOCK; ++extra) {
        block_size[extra] = 0;
    }

    int pos = 0;
    end[0] = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        size_t iC = vertexCluster[iV];
        for (std::pair<size_t, double> elem : weights[iV]) {
            neighbors[pos] = clusterLocalIndices[iC][elem.first];
            cotans[pos] = elem.second;
            pos += 1;
        }
        end[iV] = pos;
    }

    printf("About to allocate memory!\n");

    // Allocate device memory
    hipMalloc((void**)&d_x,         sizeof(double) * N);
    hipMalloc((void**)&d_b,         sizeof(double) * N);
    hipMalloc((void**)&d_Ap,        sizeof(double) * N);
    hipMalloc((void**)&d_p,         sizeof(double) * N);
    hipMalloc((void**)&d_r,         sizeof(double) * N);
    hipMalloc((void**)&d_m,         sizeof(double) * N);
    hipMalloc((void**)&d_neighbors, sizeof(int   ) * nEdges);
    hipMalloc((void**)&d_cotans,    sizeof(double) * nEdges);
    hipMalloc((void**)&d_end,       sizeof(int   ) * (N+1));
    hipMalloc((void**)&d_alpha,     sizeof(double) * 1);
    hipMalloc((void**)&d_beta,      sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,       sizeof(double) * 1);
    hipMalloc((void**)&d_local_indices, sizeof(int) * nVerticesWithDupes);
    hipMalloc((void**)&d_local_ind_end, sizeof(int) * (NBLOCK+1));
    hipMalloc((void**)&d_block_size,    sizeof(int) * NBLOCK);

    // Transfer data from host to device memory
    hipMemcpy(d_x,         x,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_r,         r,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_b,         b,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_m,         m,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors, sizeof(int   ) * nEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_end,       end,       sizeof(int   ) * (N+1),  hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,    cotans,    sizeof(double) * nEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_local_indices, local_indices, sizeof(int) * nVerticesWithDupes, hipMemcpyHostToDevice);
    hipMemcpy(d_local_ind_end, local_ind_end, sizeof(int) * (NBLOCK+1),         hipMemcpyHostToDevice);
    hipMemcpy(d_block_size,    block_size,    sizeof(int) * NBLOCK,             hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);


    printf("About to multiply matrix!\n");
    computeApClusteredCSR<<<NBLOCK,NTHREAD,8*MAX_CLUSTER_SIZE>>>(
            d_x, d_b, d_cotans, d_neighbors, d_end, d_local_indices,
            d_local_ind_end, d_block_size, d_m, t, N);

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[i] = x[i];
    }
    return 0;

    computeApClusteredCSR<<<NBLOCK,NTHREAD,8*MAX_CLUSTER_SIZE>>>(
            d_Ap, d_x, d_cotans, d_neighbors, d_end, d_local_indices,
            d_local_ind_end, d_block_size, d_m, t, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
        computeApClusteredCSR<<<NBLOCK,NTHREAD,8*MAX_CLUSTER_SIZE>>>(
                d_Ap, d_p, d_cotans, d_neighbors, d_end, d_local_indices,
                d_local_ind_end, d_block_size, d_m, t, N);
        hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
        div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
        update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
        hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
        div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
        update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
        vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 300);
    }
    if (norm >= tol)
        printf("timed out csr :'(");
    hipblasDestroy(handle);

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[i] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_m);
    hipFree(d_neighbors);
    hipFree(d_cotans);
    hipFree(d_end);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_pAp);
    hipFree(d_local_indices);
    hipFree(d_local_ind_end);
    hipFree(d_block_size);

    // Deallocate host memory
    free(x);
    free(b);
    free(r);
    free(m);
    free(cotans);
    free(neighbors);
    free(end);

    return iter * substeps;
}
