#include "hip/hip_runtime.h"
#include "cg.cuh"

#define NTHREAD 256
#define NBLOCK 500

// Computes out = (M + tL)p
__global__ void computeAp(double *out, double *p, double *cotans, int* neighbors, double* m, double t, int meshStride, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(int i = index; i < n; i += stride){
        out[i] = m[i] * p[i];
        for (int iN = 0; iN < meshStride; ++iN) {
            int neighbor = neighbors[i * meshStride + iN];
            double weight = cotans[i * meshStride + iN];
            out[i] += t * weight * (p[i] - p[neighbor]);
        }
    }
}

// Computes out = (M + tL)p
__global__ void computeApCSR(double *out, double *p, double *cotans, int* neighbors, double* m, int* end, double t, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = m[i] * p[i];
        for (int iN = end[i-1]; iN < end[i]; ++iN) {
            int neighbor = neighbors[iN];
            double weight = cotans[iN];
            out[i] += t * weight * (p[i] - p[neighbor]);
        }
    }
}

// Computes out = (M + tL)p
// TODO: make sure there are enough blocks
extern __shared__ float localArray[];
__global__ void computeApClusteredCSR(double *out,
                                      double *global_p,
                                      double *global_cotans,
                                      int* global_neighbors,
                                      int* global_vertex_end,
                                      int* global_cluster_neighbors,
                                      int* global_cluster_neighbors_end,
                                      int* cluster_end,
                                      double* m,
                                      double t,
                                      int n) {

    //int stride = gridDim.x * blockDim.x;

    int cluster_starting_vert = cluster_end[2 * blockIdx.x + 0];
    int cluster_starting_edge = cluster_end[2 * blockIdx.x + 1];
    int cluster_ending_vert   = cluster_end[2 * blockIdx.x + 2];
    int cluster_ending_edge   = cluster_end[2 * blockIdx.x + 3];

    int verts_in_cluster = cluster_ending_vert - cluster_starting_vert;
    int edges_in_cluster = cluster_ending_edge - cluster_starting_edge;
    int neighbors_of_cluster = global_cluster_neighbors_end[blockIdx.x + 1] - global_cluster_neighbors_end[blockIdx.x];

    double* cotans    = (double*)  localArray;
    int*    neighbors = (int*)    &localArray[8 * edges_in_cluster];
    int*    end       = (int*)    &localArray[8 * edges_in_cluster + 4 * edges_in_cluster];
    double* p         = (double*) &localArray[8 * edges_in_cluster + 4 * edges_in_cluster + 4 * verts_in_cluster];

    if (threadIdx.x == 0) {
        for (int iE = 0; iE < edges_in_cluster; ++iE) {
            cotans[iE]    = global_cotans[cluster_starting_edge + iE];
            neighbors[iE] = global_neighbors[cluster_starting_edge + iE];
        }

        for (int iV = 0; iV < verts_in_cluster; ++iV) {
            p[iV] = global_p[cluster_starting_vert + iV];
        }

        for (int iN = 0; iN < neighbors_of_cluster; ++iN) {
            p[verts_in_cluster + iN] = global_cluster_neighbors[global_cluster_neighbors_end[blockIdx.x] + iN];
        }
    }

    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = m[i] * p[i];
    for (int iN = end[i-1]; iN < end[i]; ++iN) {
        int neighbor = neighbors[iN];
        double weight = cotans[iN];
        out[i] += t * weight * (p[i] - p[neighbor]);
    }
}

// Computes out = a-b
__global__ void vector_sub(double *out, double *a, double *b, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i] - b[i];
  }
}

// Copies a into out
__global__ void vector_cpy(double *out, double *a, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i];
  }
}

// Computes out = num / denom
__global__ void div(double *out, double *num, double *denom) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0) {
      *out = *num / *denom;
  }
}

// x += alpha p
// r -= alpha Ap
__global__ void update_x_r(double* x, double* r, double* alpha, double* p, double* Ap, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    x[i] += alpha[0] * p[i];
    r[i] -= alpha[0] * Ap[i];
  }
}

// Computes p = beta * p + r
__global__ void update_p(double* p, double *beta, double *r, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    p[i] = beta[0] * p[i] + r[i];
  }
}

// For each vertex, returns a map which maps the indices of the vertex's neighbors to the
// entries of the laplacian corresponding to those edges
std::vector<std::unordered_map<size_t, double>> edgeWeights(const TetMesh& mesh) {
    std::vector<std::unordered_map<size_t, double>> weights;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        std::unordered_map<size_t, double> vWeights;
        weights.push_back(vWeights);
    }

    for (size_t iPE = 0; iPE < mesh.edges.size(); ++iPE) {
        PartialEdge pe = mesh.edges[iPE];
        size_t vSrc    = pe.src;
        size_t vDst    = pe.dst;

        double weight = mesh.partialEdgeCotanWeights[iPE];
        weights[vSrc][vDst] += weight;
        weights[vDst][vSrc] += weight;
    }

    return weights;
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
int  cgSolve(Eigen::VectorXd& xOut, Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose) {
    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors;
    int N = bVec.size();

    int maxDegree = 0;
    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        maxDegree = std::max(maxDegree, (int) weights[iV].size());
    }

    // Allocate host memory
    x         = (double*) malloc(sizeof(double) * N);
    b         = (double*) malloc(sizeof(double) * N);
    r         = (double*) malloc(sizeof(double) * N);
    m         = (double*) malloc(sizeof(double) * N);
    cotans    = (double*) malloc(sizeof(double) * N * maxDegree);
    neighbors = (int*  ) malloc(sizeof(int  ) * N * maxDegree);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[i];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[i];
    }
    if (t < 0) t = 1;

    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        size_t neighborCount = 0;
        for (std::pair<size_t, double> elem : weights[iV]) {
            neighbors[iV * maxDegree + neighborCount] = elem.first;
            cotans[iV * maxDegree + neighborCount] = elem.second;
            ++neighborCount;
        }

        // Fill in the remaining slots with zeros
        for (size_t iN = neighborCount; iN < maxDegree; ++iN) {
            neighbors[iV * maxDegree + iN] = iV;
            cotans[iV * maxDegree + iN] = 0;
        }
    }

    //printf("max degree: %d\n", maxDegree);
    Eigen::SparseMatrix<double> L    = mesh.weakLaplacian();
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        for (size_t iN = 0; iN < maxDegree; ++iN) {
            size_t jV = neighbors[iV * maxDegree + iN];
            if (jV == iV) continue;

            double mat = L.coeffRef(iV, jV);
            double arr = cotans[iV * maxDegree + iN];
            if (abs(mat + arr) >= 1e-7) {
                printf("ERROR: matrix is %f\tarray is %f\terror is %.10e\n",
                        mat, arr, abs(mat + arr));
            }
        }
    }

    // Allocate device memory
    hipMalloc((void**)&d_x,         sizeof(double) * N);
    hipMalloc((void**)&d_b,         sizeof(double) * N);
    hipMalloc((void**)&d_Ap,        sizeof(double) * N);
    hipMalloc((void**)&d_p,         sizeof(double) * N);
    hipMalloc((void**)&d_r,         sizeof(double) * N);
    hipMalloc((void**)&d_m,         sizeof(double) * N);
    hipMalloc((void**)&d_neighbors, sizeof(int   ) * N * maxDegree);
    hipMalloc((void**)&d_cotans,    sizeof(double) * N * maxDegree);
    hipMalloc((void**)&d_alpha,     sizeof(double) * 1);
    hipMalloc((void**)&d_beta,      sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,       sizeof(double) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_x,         x,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_r,         r,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_b,         b,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_m,         m,         sizeof(double) * N,             hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors, sizeof(int   ) * N * maxDegree, hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,    cotans,    sizeof(double) * N * maxDegree, hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE); 

    computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_x, d_cotans, d_neighbors, d_m, t, maxDegree, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
         computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_p, d_cotans, d_neighbors, d_m, t, maxDegree, N);
         hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
         div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
         update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
         hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
         div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
         update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
         vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 300);
    }
    hipblasDestroy(handle);

    if (norm >= tol)
        printf("timed out :'(");

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[i] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_neighbors);
    hipFree(d_cotans);

    // Deallocate host memory
    free(x);
    free(b);
    free(cotans);
    free(neighbors);

    return iter * substeps;
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
// Stores matrix in CSR format
int cgSolveCSR(Eigen::VectorXd& xOut, Eigen::VectorXd bVec, const TetMesh& mesh, double tol, double t, bool verbose) {
    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors, *end, *d_end;
    int N = bVec.size();

    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    int nEdges = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        nEdges += weights[iV].size();
    }

    // Allocate host memory
    x         = (double*) malloc(sizeof(double) * N);
    b         = (double*) malloc(sizeof(double) * N);
    r         = (double*) malloc(sizeof(double) * N);
    m         = (double*) malloc(sizeof(double) * N);
    cotans    = (double*) malloc(sizeof(double) * nEdges);
    neighbors = (int*   ) malloc(sizeof(int   ) * nEdges);
    end = (int*   ) malloc(sizeof(int   ) * 2 * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[i];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[i];
    }
    if (t < 0) t = 1;

    int pos = 0;
    end[0] = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        for (std::pair<size_t, double> elem : weights[iV]) {
            neighbors[pos] = elem.first;
            cotans[pos] = elem.second;
            pos += 1;
        }
        end[iV] = pos;
    }

    // Allocate device memory
    hipMalloc((void**)&d_x,         sizeof(double) * N);
    hipMalloc((void**)&d_b,         sizeof(double) * N);
    hipMalloc((void**)&d_Ap,        sizeof(double) * N);
    hipMalloc((void**)&d_p,         sizeof(double) * N);
    hipMalloc((void**)&d_r,         sizeof(double) * N);
    hipMalloc((void**)&d_m,         sizeof(double) * N);
    hipMalloc((void**)&d_neighbors, sizeof(int   ) * nEdges);
    hipMalloc((void**)&d_cotans,    sizeof(double) * nEdges);
    hipMalloc((void**)&d_end,       sizeof(int   ) * (N+1));
    hipMalloc((void**)&d_alpha,     sizeof(double) * 1);
    hipMalloc((void**)&d_beta,      sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,    sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,       sizeof(double) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_x,         x,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_r,         r,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_b,         b,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_m,         m,         sizeof(double) * N,      hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors, neighbors, sizeof(int   ) * nEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_end,       end,       sizeof(int   ) * (N+1),  hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,    cotans,    sizeof(double) * nEdges, hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    computeApCSR<<<NBLOCK,NTHREAD>>>(d_Ap, d_x, d_cotans, d_neighbors, d_m, d_end, t, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
        computeApCSR<<<NBLOCK,NTHREAD>>>(d_Ap, d_p, d_cotans, d_neighbors, d_m, d_end, t, N);
        hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
        div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
        update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
        hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
        div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
        update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
        vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 300);
    }
    if (norm >= tol)
        printf("timed out :'(");
    hipblasDestroy(handle);

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
      xOut[i] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_neighbors);
    hipFree(d_end);
    hipFree(d_cotans);

    // Deallocate host memory
    free(x);
    free(b);
    free(cotans);
    free(neighbors);

    return iter * substeps;
}

// If t < 0, solve Lx = b (realy we relax to (L + 1e-12)x = b to ensure our
// system is positive definite
// If t >= 0, solve (M + tL)x = b, where M is the mass matrix and L is the laplacian
// Stores matrix in CSR format
int cgSolveClusteredCSR(Eigen::VectorXd& xOut, Eigen::VectorXd& bVec, const TetMesh& mesh, double tol, double t, bool verbose) {
    double *x, *b, *r, *cotans, *m;
    double *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_old_r2, *d_new_r2,  *d_pAp, *d_alpha, *d_beta, *d_cotans, *d_m;
    int* neighbors, *d_neighbors, *vertex_end, *d_vertex_end, *cluster_end, *d_cluster_end;
    int  *cluster_neighbors, *d_cluster_neighbors, *cluster_neighbors_end, *d_cluster_neighbors_end;
    int N = bVec.size();

    printf("Started again\n");

    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    int nEdges = 0;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        nEdges += weights[iV].size();
    }

    size_t clusterSize = 256;
    std::vector<std::vector<size_t>> clusters, clusterNeighbors;
    std::tie(clusters, clusterNeighbors) = clusterAndNeighbors(mesh, clusterSize);
    size_t nClusters = clusters.size();
    std::vector<size_t> vertexReordering(N, 0);        // vertexReordering[position in big array] = iV
    //std::vector<size_t> inverseVertexReordering(N, 0); // inverseVertexReordering[iV] = position in big array
    std::vector<std::unordered_map<size_t, size_t>> localInverseReordering;

    size_t count = 0;
    size_t nNeighbors = 0;
    for (size_t iC = 0; iC < nClusters; ++iC) {
        std::vector<size_t> cluster = clusters[iC];
        std::unordered_map<size_t, size_t> clusterInverseMap;
        size_t localCount = 0;
        for (size_t iV : cluster) {
            vertexReordering[count] = iV;
            clusterInverseMap[iV] = localCount;
            //inverseVertexReordering[iV] = count;
            count++;
            localCount++;
        }
        for (size_t iN : clusterNeighbors[iC]) {
            clusterInverseMap[iN] = localCount;
            localCount++;
            nNeighbors++;
        }
        localInverseReordering.push_back(clusterInverseMap);
    }

    // Allocate host memory
    x           = (double*) malloc(sizeof(double) * N);
    b           = (double*) malloc(sizeof(double) * N);
    r           = (double*) malloc(sizeof(double) * N);
    m           = (double*) malloc(sizeof(double) * N);
    cotans      = (double*) malloc(sizeof(double) * nEdges);
    neighbors   = (int*   ) malloc(sizeof(int   ) * nEdges);
    vertex_end  = (int*   ) malloc(sizeof(int   ) * (N+nClusters));
    cluster_end = (int*   ) malloc(sizeof(int   ) * 2 * (nClusters+1));
    cluster_neighbors     = (int*) malloc(sizeof(int) * nNeighbors);
    cluster_neighbors_end = (int*) malloc(sizeof(int) * (nClusters+1));

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.0f;
        b[i] = bVec[vertexReordering[i]];
        r[i] = b[i];
        m[i] = (t < 0)?1e-12:mesh.vertexDualVolumes[vertexReordering[i]];
    }
    if (t < 0) t = 1;
    return 0;

    //double* cotans    = (double*)  localArray;
    //int*    neighbors = (int*)    &localArray[8 * edges_in_cluster];
    //int*    end       = (int*)    &localArray[8 * edges_in_cluster + 4 * edges_in_cluster];
    //double* p         = (double*) &localArray[8 * edges_in_cluster + 4 * edges_in_cluster + 4 * verts_in_cluster];
    // Data per block: edges_in_cluster doubles + edges_in_cluster ints + verts_in_cluster ints + verts_accessed_by_cluster doubles
    // = 12 * edges_in_cluster + 4 * verts_in_cluster + 8 * verts_accessed_by_cluster


    // Cluster end = [vertex end, edge end, vertex end, edge end, ...
    // Vertex end is in local indices
    size_t MAX_CLUSTER_DATA_SIZE = 0;
    int pos = 0;
    int vIdx = 0;
    int neighborPos = 0;
    cluster_end[0] = 0;
    cluster_end[1] = 0;
    cluster_neighbors_end[0] = 0;
    for (size_t iC = 0; iC < nClusters; ++iC) {
        std::vector<size_t> cluster = clusters[iC];
        int localPos = 0;
        int edges_in_cluster = 0;
        vertex_end[pos] = 0;
        for (size_t iV : cluster) {
            for (std::pair<size_t, double> elem : weights[iV]) {
                neighbors[pos] = localInverseReordering[iC][elem.first];
                cotans[pos] = elem.second;
                pos += 1;
                localPos += 1;
                edges_in_cluster += 1;
            }
            vertex_end[vIdx + iC + 1] = localPos;
            vIdx += 1;
        }
        cluster_end[2 * iC + 2] = vIdx;
        cluster_end[2 * iC + 3] = pos;

        for (size_t iN : clusterNeighbors[iC]) {
            cluster_neighbors[neighborPos] = iN;
            neighborPos++;
        }
        cluster_neighbors_end[iC+1] = neighborPos;
        MAX_CLUSTER_DATA_SIZE = std::max(MAX_CLUSTER_DATA_SIZE,
                12 * edges_in_cluster + 12 * cluster.size() + 8 * clusterNeighbors[iC].size());
    }
    printf("MAX_CLUSTER_DATA_SIZE: %d\n", (int) MAX_CLUSTER_DATA_SIZE);
    return 0;

    // Allocate device memory
    hipMalloc((void**)&d_x,           sizeof(double) * N);
    hipMalloc((void**)&d_b,           sizeof(double) * N);
    hipMalloc((void**)&d_Ap,          sizeof(double) * N);
    hipMalloc((void**)&d_p,           sizeof(double) * N);
    hipMalloc((void**)&d_r,           sizeof(double) * N);
    hipMalloc((void**)&d_m,           sizeof(double) * N);
    hipMalloc((void**)&d_neighbors,   sizeof(int   ) * nEdges);
    hipMalloc((void**)&d_cotans,      sizeof(double) * nEdges);
    hipMalloc((void**)&d_vertex_end,  sizeof(int   ) * (N+nClusters));
    hipMalloc((void**)&d_cluster_end, sizeof(int   ) * 2 * (nClusters+1));
    hipMalloc((void**)&d_alpha,       sizeof(double) * 1);
    hipMalloc((void**)&d_beta,        sizeof(double) * 1);
    hipMalloc((void**)&d_old_r2,      sizeof(double) * 1);
    hipMalloc((void**)&d_new_r2,      sizeof(double) * 1);
    hipMalloc((void**)&d_pAp,         sizeof(double) * 1);
    hipMalloc((void**)&d_cluster_neighbors,     sizeof(int   ) * nNeighbors);
    hipMalloc((void**)&d_cluster_neighbors_end, sizeof(int   ) * (nClusters+1));

    // Transfer data from host to device memory
    hipMemcpy(d_x,           x,           sizeof(double) * N,                 hipMemcpyHostToDevice);
    hipMemcpy(d_r,           r,           sizeof(double) * N,                 hipMemcpyHostToDevice);
    hipMemcpy(d_b,           b,           sizeof(double) * N,                 hipMemcpyHostToDevice);
    hipMemcpy(d_m,           m,           sizeof(double) * N,                 hipMemcpyHostToDevice);
    hipMemcpy(d_neighbors,   neighbors,   sizeof(int   ) * nEdges,            hipMemcpyHostToDevice);
    hipMemcpy(d_vertex_end,  vertex_end,  sizeof(int   ) * (N+nClusters),     hipMemcpyHostToDevice);
    hipMemcpy(d_cluster_end, cluster_end, sizeof(int   ) * 2 * (nClusters+1), hipMemcpyHostToDevice);
    hipMemcpy(d_cotans,      cotans,      sizeof(double) * nEdges,            hipMemcpyHostToDevice);
    hipMemcpy(d_cluster_neighbors,     cluster_neighbors,     sizeof(int) * nNeighbors,     hipMemcpyHostToDevice);
    hipMemcpy(d_cluster_neighbors_end, cluster_neighbors_end, sizeof(int) * (nClusters+1),  hipMemcpyHostToDevice);

    bool done = false;
    int iter = 0;


    computeApClusteredCSR<<<NBLOCK,NTHREAD, MAX_CLUSTER_DATA_SIZE>>>(d_Ap, d_b, d_cotans, d_neighbors, d_vertex_end,
                                     d_cluster_neighbors, d_cluster_neighbors_end, d_cluster_end, d_m, t, N);
    // Transfer data back to host memory
    hipMemcpy(b, d_Ap, sizeof(double) * N, hipMemcpyDeviceToHost);

    printf("here\n");
    for (int i = 0; i < N; ++i) {
      bVec[vertexReordering[i]] = b[i];
    }
    printf("there\n");

    // https://stackoverflow.com/questions/12400477/retaining-dot-product-on-gpgpu-using-cublas-routine/12401838#12401838
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    //__global__ void computeApClusteredCSR(double *out,
                                      //double *global_p,
                                      //double *global_cotans,
                                      //int* global_neighbors,
                                      //int* global_vertex_end,
                                      //int* global_cluster_neighbors,
                                      //int* global_cluster_neighbors_end,
                                      //int* cluster_end,
                                      //double* m,
                                      //double t,
                                      //int n) {

    computeApClusteredCSR<<<NBLOCK,NTHREAD, MAX_CLUSTER_DATA_SIZE>>>(d_Ap, d_x, d_cotans, d_neighbors, d_vertex_end,
                                     d_cluster_neighbors, d_cluster_neighbors_end, d_cluster_end, d_m, t, N);
    vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);
    hipblasDdot(handle, N, d_r, 1, d_r, 1, d_old_r2);

    int substeps = 40;
    double norm = 0;
    while (!done) {
      for (int i = 0; i < substeps; ++i) {
        computeApClusteredCSR<<<NBLOCK,NTHREAD, MAX_CLUSTER_DATA_SIZE>>>(d_Ap, d_p, d_cotans, d_neighbors, d_vertex_end,
                                         d_cluster_neighbors, d_cluster_neighbors_end, d_cluster_end, d_m, t, N);
        hipblasDdot(handle, N, d_p, 1, d_Ap, 1, d_pAp);
        div<<<NBLOCK, NTHREAD>>>(d_alpha, d_old_r2, d_pAp);
        update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
        hipblasDdot(handle, N, d_r, 1, d_r, 1, d_new_r2);
        div<<<NBLOCK, NTHREAD>>>(d_beta, d_new_r2, d_old_r2);
        update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
        vector_cpy<<<NBLOCK,NTHREAD>>>(d_old_r2, d_new_r2, 1);
      }

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(double) * N, hipMemcpyDeviceToHost);
      norm = 0;
      for (int i = 0; i < N; i++) {
        norm = fmax(norm, fabs(r[i]));
      }
      ++iter;
      if (verbose) printf("%d: residual: %f\n", iter, norm);
      done = (norm < tol) || (iter > 300);
    }
    if (norm >= tol)
        printf("timed out :'(");
    printf("DOne solving\n");
    hipblasDestroy(handle);

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost);

    printf("here\n");
    for (int i = 0; i < N; ++i) {
      xOut[vertexReordering[i]] = x[i];
    }
    printf("there\n");

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_old_r2);
    hipFree(d_new_r2);
    hipFree(d_neighbors);
    hipFree(d_cotans);

    // Deallocate host memory
    free(x);
    free(b);
    free(cotans);
    free(neighbors);

    printf("returning\n");
    return 0;
}
