#include "hip/hip_runtime.h"
#include "cg.cuh"

#define NTHREAD 1
#define NBLOCK  1

__global__ void computeAp(float *out, float *p, float *cotans, int* neighbors, int meshStride, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(int i = index; i < n; i += stride){
        out[i] = 1e-12 * p[i];
        for (int iN = 0; iN < meshStride; ++iN) {
            int neighbor = neighbors[iN];
            double weight = cotans[iN];
            out[i] += weight * (p[i] - p[neighbor]);
        }
    }
}

__global__ void vector_sub(float *out, float *a, float *b, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i] - b[i];
  }
}

__global__ void vector_cpy(float *out, float *a, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    out[i] = a[i];
  }
}

__global__ void compute_alpha(float *out, float *r2, float *r, float *p, float *Ap, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0) {
    r2[0] = 0;
    float denom = 0;
    for (int i = 0; i < n; i += 1) {
      r2[0] += r[i] * r[i];
      denom += p[i] * Ap[i];
    }
    out[0] = r2[0] / denom;
  }
}

__global__ void update_x_r(float* x, float* r, float* alpha, float* p, float* Ap, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
    x[i] += alpha[0] * p[i];
    r[i] -= alpha[0] * Ap[i];
  }
}

__global__ void compute_beta(float *out, float *r2, float *r, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0) {
    out[0] = 1/r2[0];
    r2[0] = 0;
    for (int i = 0; i < n; i += 1) {
      r2[0] += r[i] * r[i];
    }
    out[0] *= r2[0];
    r2[0] = 5;
  }
}

__global__ void set_r2(float *r2) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0) {
    r2[0] = 5;
  }
}

// Returns p = beta * p + r
__global__ void update_p(float* p, float *beta, float *r, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < n; i += stride){
      p[i] *= beta[0];
      p[i] += r[i];
    //p[i] = beta[0] * p[i] + r[i];
  }
}

std::vector<std::unordered_map<size_t, double>> edgeWeights(const TetMesh& mesh) {
    std::vector<std::unordered_map<size_t, double>> weights;
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        std::unordered_map<size_t, double> vWeights;
        weights.push_back(vWeights);
    }

    for (size_t iPE = 0; iPE < mesh.edges.size(); ++iPE) {
        PartialEdge pe = mesh.edges[iPE];
        size_t vSrc    = pe.src;
        size_t vDst    = pe.dst;

        double weight = mesh.partialEdgeCotanWeights[iPE];

        auto findSrc = weights[vSrc].find(vDst);
        if (findSrc == weights[vSrc].end()) {
            weights[vSrc][vDst] = weight;
        } else {
            findSrc->second += weight;
        }

        auto findDst = weights[vDst].find(vSrc);
        if (findDst == weights[vDst].end()) {
            weights[vDst][vSrc] = weight;
        } else {
            findDst->second += weight;
        }

    }

    return weights;
}

void cgSolve(Eigen::VectorXd& xOut, Eigen::VectorXd bVec, const TetMesh& mesh, double t) {
    float *x, *b, *r;
    float *d_x, *d_b, *d_p, *d_Ap, *d_r, *d_r2, *d_alpha, *d_beta;
    int N = bVec.size();

    // Allocate host memory
    x   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    r   = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 0.13f;
        b[i] = bVec[i];
        r[i] = b[i];
        if (abs(b[i]) > 0) {
            printf("\tBIG B: i = %d, \t b = %f\n", i, b[i]);
        }
    }

    // Allocate device memory
    hipMalloc((void**)&d_x,     sizeof(float) * N);
    hipMalloc((void**)&d_b,     sizeof(float) * N);
    hipMalloc((void**)&d_Ap,    sizeof(float) * N);
    hipMalloc((void**)&d_p,     sizeof(float) * N);
    hipMalloc((void**)&d_r,     sizeof(float) * N);
    hipMalloc((void**)&d_alpha, sizeof(float) * 1);
    hipMalloc((void**)&d_beta,  sizeof(float) * 1);
    hipMalloc((void**)&d_r2,    sizeof(float) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_x, x, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel

    bool done = false;
    int iter = 0;

    float *cotans;
    int *neighbors;
    int maxDegree = 0;
    std::vector<std::unordered_map<size_t, double>> weights = edgeWeights(mesh);
    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        maxDegree = std::max(maxDegree, (int) weights[iV].size());
    }

    cotans    = (float*) malloc(sizeof(float) * maxDegree * N);
    neighbors = (int*)   malloc(sizeof(int)   * maxDegree * N);

    for (size_t iV = 0; iV < mesh.vertices.size(); ++iV) {
        size_t neighborCount = 0;
        for (std::pair<size_t, double> elem : weights[iV]) {
            neighbors[iV * maxDegree + neighborCount] = elem.first;
            cotans[iV * maxDegree + neighborCount] = elem.second;
            ++neighborCount;
        }

        // Fill in the remaining slots with zeros
        for (size_t iN = neighborCount; iN < maxDegree; ++iN) {
            neighbors[iV * maxDegree + iN] = iV;
            cotans[iV * maxDegree + iN] = 0;
        }
    }
    printf("max degree: %d\n", maxDegree);

    //computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_x, cotans, neighbors, maxDegree, N);
    //vector_sub<<<NBLOCK,NTHREAD>>>(d_r, d_b, d_Ap, N);
    //vector_cpy<<<NBLOCK,NTHREAD>>>(d_p, d_r, N);

    float *Ap = (float*)malloc(sizeof(float) * N);
    hipMemcpy(r, d_r, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(Ap, d_Ap, sizeof(float) * N, hipMemcpyDeviceToHost);
    printf("r[0] : %f \t r[1] : %f \t r[2] : %f\n", r[0], r[1], r[2]);
    printf("b[0] : %f \t b[1] : %f \t b[2] : %f\n", b[0], b[1], b[2]);
    printf("Ap[0] : %f \t Ap[1] : %f \t Ap[2] : %f\n", Ap[0], Ap[1], Ap[2]);
    printf("\n");
    while (!done) {
      for (int i = 0; i < 1; ++i) {
        // computeAp<<<NBLOCK,NTHREAD>>>(d_Ap, d_p, cotans, neighbors, maxDegree, N);
        // compute_alpha<<<NBLOCK, NTHREAD>>>(d_alpha, d_r2, d_r, d_p, d_Ap, N);
        // update_x_r<<<NBLOCK, NTHREAD>>>(d_x, d_r, d_alpha, d_p, d_Ap, N);
        compute_beta<<<NBLOCK, NTHREAD>>>(d_beta, d_r2, d_r, N);
        update_p<<<NBLOCK, NTHREAD>>>(d_p, d_beta, d_r, N);
      }
      compute_beta<<<NBLOCK, NTHREAD>>>(d_beta, d_r2, d_r, N);

      // Transfer data back to host memory
      hipMemcpy(r, d_r, sizeof(float) * N, hipMemcpyDeviceToHost);

      float *alpha = new float[1];
      //float *beta = new float[1];
      float *r2 = (float*)malloc(sizeof(float));
      //float *p = new float[N];
      //hipMemcpy(p,     d_p,     sizeof(float) * N, hipMemcpyDeviceToHost);
      //hipMemcpy(x,     d_x,     sizeof(float) * N, hipMemcpyDeviceToHost);
      hipMemcpy(r2,    d_r2,    sizeof(float) * 1, hipMemcpyDeviceToHost);
      hipMemcpy(alpha, d_alpha, sizeof(float) * 1, hipMemcpyDeviceToHost);
      //hipMemcpy(beta,  d_beta,  sizeof(float) * 1, hipMemcpyDeviceToHost);
      //float norm = 0;
      //for (int i = 0; i < N; i++) {
        //norm = fmax(norm, r[i] * r[i]);
      //}
      ++iter;
      printf("alpha: %f \t r^2  : %f\n", alpha[0], r2[0]);
      //printf("x[0] : %f \t x[1] : %f \t x[2] : %f\n", x[0], x[1], x[2]);
      //printf("p[0] : %f \t p[1] : %f \t p[2] : %f\n", p[0], p[1], p[2]);
      printf("r[0] : %f \t r[1] : %f \t r[2] : %f\n", r[0], r[1], r[2]);
      //printf("norm: %f\n", norm);
      //printf("\n");
      //fflush(stdout);
      //done = (norm < 1e-4) || (iter > 0);
      done = true;
    }

    // Transfer data back to host memory
    hipMemcpy(x, d_x, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; ++i) {
      float result = 1e-12;
      for (int iN = 0; iN < maxDegree; ++iN) {
          int neighbor = neighbors[iN];
          double weight = cotans[iN];
          result += weight * (x[i] - x[neighbor]);
      }
      if (abs(result - b[i]) > 1e-4) {
          printf("err: vertex %d result[%d] = %f, b[%d] = %f, x[%d] = %f\n", i, i, result, i, b[i], i, x[i]);
          printf("iter: %d\n", iter);
      }
      xOut[i] = x[i];
    }

    // Deallocate device memory
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Ap);
    hipFree(d_p);
    hipFree(d_r);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipFree(d_r2);

    // Deallocate host memory
    free(x);
    free(b);
    free(cotans);
    free(neighbors);

    return;
}
